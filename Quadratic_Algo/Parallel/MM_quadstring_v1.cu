#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include<time.h>
#include<math.h>
#include <stdbool.h>
#include <sys/resource.h>
#include <errno.h>
#include <unistd.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 32
#define MAX 10
#define MIN 1
#define ll long long int


//Return the number of digits in the num
int num_digits(ll num){
    int len = 0;
    int n = (int)num;
    while(n > 0){
        len++;
        n = n/10;
    }
    return len;
}


//N is the number of rows
__global__ void matrix_simp1(int *a, char* temp_a, int M, int P, int N) {         
    int starting =0;

    int idx =  blockDim.x * blockIdx.x + threadIdx.x;
    
    char arr[10];
    if (idx < N) {
        for(int j=0; j<N; j++){
            ll num = a[idx*N+j];

            //Calculating length of a num
            int len = 0;
            int no = (int)num;
            while(no > 0){
                len++;
                no = no/10;
            }
            int digit = len;

            
            // char* arr1;
            char arr1[5];
            // cudaMalloc((void**)&arr1, digit*sizeof(char));
            if(j!=0){
                for(int z=0;z<(P-digit);z++){
                    arr[starting+z] = '0';
                }
                starting += (P-digit);
            }
            int index = 0;
            while (num) {
                // Separate last digit from
                // the number and add ASCII
                // value of character '0' is 48
                arr1[++index] = num % 10 + '0';

                // Truncate the last
                // digit from the number
                num /= 10;
            }
            // Reverse the array for result
            for (int z = 0; z < index; z++) {
                arr[starting+z] = arr1[index - z];
            }
            starting += digit;
        }
        arr[starting] = '\0';
        printf("arr: %s\n",arr);
        for(int i=0;i<starting;i++){
                temp_a[idx*(N+1)*P+i] = arr[i];
        }
        
    }

    // free(arr);

    return;
}

__global__ void matrix_simp2(int *b, char* temp_b, int M, int P, int N) { 
    int starting =0;

    int idx =  blockDim.x * blockIdx.x + threadIdx.x;
    
    char arr[10];
    if (idx < N) {

        for(int j=0; j<N; j++){
            ll num = b[(N-1-j)*N+idx];

            //Calculating length of a num
            int len = 0;
            int no = (int)num;
            while(no > 0){
                len++;
                no = no/10;
            }
            int digit = len;

            
            // char* arr1;
            char arr1[5];
            // cudaMalloc((void**)&arr1, digit*sizeof(char));
            if(j!=0){
                for(int z=0;z<(P-digit);z++){
                    arr[starting+z] = '0';
                }
                starting += (P-digit);
            }
            int index = 0;
            while (num) {
                // Separate last digit from
                // the number and add ASCII
                // value of character '0' is 48
                arr1[++index] = num % 10 + '0';

                // Truncate the last
                // digit from the number
                num /= 10;
            }
            // Reverse the array for result
            for (int z = 0; z < index; z++) {
                arr[starting+z] = arr1[index - z];
            }
            starting += digit;
        }
        arr[starting] = '\0';
        printf("arr: %s\n",arr);
        for(int i=0;i<starting;i++){
            // if(arr[i]!='\0')
                temp_b[idx*(N+1)*P+i] = arr[i];  
        }
        
    }

    return;
}

__global__ void matrix_mul(char* temp_a, char* temp_b, int* temp_c, int P, int N){
    //rows
    int idx_i = blockDim.x * blockIdx.x + threadIdx.x; 
    //cols
    int idx_j = blockDim.y * blockIdx.y + threadIdx.y; 

    int num_size= (N+1)*P;

    if(idx_i < N && idx_j < N){
        //find the last index of the number
        int l1=0,l2=0;
        for(int i=0;i<num_size;i++){
            if(temp_a[idx_i*num_size + i]=='\u0000') break;
            else l1++;
        }
        for(int i=0;i<num_size;i++){
            if(temp_b[idx_j*num_size + i]=='\u0000') break;
            else l2++;
        }
        
        printf("%d %d %d %d\n",idx_i,idx_j,l1,l2);

        for(int i=0;i<l2;i++){
            for(int j=0;j<l1;j++){
                //a will start from ones place, ||ly b
                int index1,index2;
                index1 = idx_i*num_size + l1 - j - 1;
                index2 = idx_j*num_size + l2 - i - 1;
                int a = temp_a[index1] - '0';
                int b = temp_b[index2] - '0';
                // printf("\na: %d , b: %d",a,b);   
                temp_c[(idx_i*N + idx_j)*(2*num_size) + i+j] += a*b;
            }
        }
        //we will handle carry overs in the CPU
    }
}
int main(){

    // struct timespec start,end;
    // double runTime;
    int n=3;
    // int N=n*n;

    int *a,*b,*c,*c1;

    struct rusage r_usage;

    //Allocating space for matrix a,b,c
    a = (int *)malloc(n*n*sizeof(int));
    b = (int *)malloc(n*n*sizeof(int));
    c = (int *)malloc(n*n*sizeof(int));
    // c1 = (int *)malloc(n*n*sizeof(int));


    int mx=0;
    // Generate random matrix a and b
    for(int i=0; i<n; i++){
        for(int j=0; j<n; j++){
            //This generates the random integer number between MIN and MAX
            a[i*n + j] = (int) (( rand() /(float) (RAND_MAX) ) * (MAX - MIN) + MIN);
            b[i*n + j] =  (int)(( rand() /(float) (RAND_MAX) ) * (MAX - MIN) + MIN);

            if(mx<a[i*n+j]) mx=a[i*n+j];
            if(mx<b[i*n+j]) mx=b[i*n+j];

            printf("%d ", a[i*n+j]);
        }
        printf("\n");
    }
    for(int i=0; i<n; i++){
        for(int j=0; j<n; j++){
            printf("%d ", b[i*n+j]);
        }
        printf("\n");
    }

    //M is the number of digits in the max elements.
    int M = num_digits(mx);
    
    //P is the possible max number of digits in the answer matrix
    int P = num_digits(n*(pow(10,2*M)-1));

    //array of string = temp_a
    char* temp_a = (char*)malloc(n*(n+1)*P*sizeof(char)); 
    char* temp_b = (char*)malloc(n*(n+1)*P*sizeof(char)); 
    int* temp_c = (int*)malloc(2*n*n*(n+1)*P*sizeof(int)); 


    //device variables
    int* d_a;
    char* d_temp_a;
    int* d_b;
    char* d_temp_b;
    int *d_temp_c;
  


    // Alloc space for device copies
    hipMalloc((void**)&d_a, n*n*sizeof(int));
    hipMalloc((void**)&d_temp_a,n*(n+1)*P*sizeof(char));
    hipMalloc((void**)&d_b, n*n*sizeof(int));
    hipMalloc((void**)&d_temp_b,n*(n+1)*P*sizeof(char));
    hipMalloc((void**)&d_temp_c,2*n*n*(n+1)*P*sizeof(int));

    

    // Copy to device
    hipMemcpy(d_a, a, n*n*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n*n*sizeof(int), hipMemcpyHostToDevice);


    printf("Before Launching the kernel:\n");
    // Launch kernel on GPU
    // int ceilB = (n+BLOCK_SIZE-1)/BLOCK_SIZE;
    // ceilB, BLOCK_SIZE
    matrix_simp1<<<1,n>>>(d_a, d_temp_a, M,P,n);
    matrix_simp2<<<1,n>>>(d_b, d_temp_b, M,P,n);
    
    // Copy result back to host
    hipMemcpy(temp_a, d_temp_a, n*(n+1)*P*sizeof(char), hipMemcpyDeviceToHost);
    hipMemcpy(temp_b, d_temp_b, n*(n+1)*P*sizeof(char), hipMemcpyDeviceToHost);
    
    
    // Print result
    printf("temp_a : ");
    // int* arr1 = (int*)malloc
    for(int i=0;i<n*(n+1)*P;i++){
        // if(temp_a[i] !='\u0000')
        printf("%c ", temp_a[i]);
    }
    
    printf("\n");
    
    printf("temp_b : ");
    for(int i=0;i<n*(n+1)*P;i++){
        printf("%c ", temp_b[i]);
    }
    
    printf("\n");
    
    /********************MULTIPLY************************/
    dim3 gridShape = dim3(1,1);
    dim3 blockShape = dim3(n,n);
    matrix_mul<<<gridShape,blockShape>>>(d_temp_a,d_temp_b,d_temp_c,P,n);
    printf("GPU process completed:\n");
    
    hipMemcpy(temp_c, d_temp_c, 2*n*n*(n+1)*P*sizeof(int), hipMemcpyDeviceToHost);
    
    printf("\ntemp_c : ");
    for(int i=0;i<2*n*n*(n+1)*P;i++){
        printf("%d ", temp_c[i]);
    }

    // Cleanup
    free(a);
    free(b); 
    free(temp_a);
    free(temp_b);
    free(temp_c);

    hipFree(d_a); 
    hipFree(d_temp_a);
    hipFree(d_b);
    hipFree(d_temp_b);
    hipFree(d_temp_c);


    return 0;
}

// 9 8 7 6 5 4 3 2 1
// 9 6 3 8 5 2 7 4 1

// 1 2 3
// 4 5 6
// 7 8 9

// a= \u0000

/********
A
8 8 9 
4 3 5 
4 9 6
B
4 8 2 
7 5 6 
5 9 7
arr: 8008009
arr: 4003005
arr: 4009006
arr: 5007004
arr: 9005008
arr: 7006002

4,00,96,13,30,95,036    7,21,12,18,51,09,072    
2,00,43,06,20,47,020

40 0 0 96 0 0 133 0 0 95 0 0 36 0 0 0 0 0 0 0 0 0 0 0
0 0 0 0 0 0 0 0 0 0 0 0 72 0 0 112 0 0 185 0 0 109 0 0 72 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0
56 0 0 104 0 0 127 0 0 70 0 0 38 0 0 43 0 0 62 0 0 47 0 0 20 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 36 0 0 47 0 0 92 0 0 49 0 0 40 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 28 0 0 45 0 0 61 0 0 36 0 0 30 0 0 73 0 0 109 0 0 78 0 0 24 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 36 0 0 101 0 0 131 0 0 102 0 0

*/